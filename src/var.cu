#include "var.cuh"

int MESH = 128;
int NX = MESH; int NY = MESH; int NZ = MESH;  

__constant__ float CSSQ;
__constant__ float OMEGA;
__constant__ float SHARP_C;
__constant__ float SIGMA;
__constant__ float W[NLINKS];
__constant__ int CIX[NLINKS], CIY[NLINKS], CIZ[NLINKS];

float *d_f, *d_g;
float *d_normx, *d_normy, *d_normz, *d_indicator;
float *d_curvature, *d_ffx, *d_ffy, *d_ffz;
float *d_ux, *d_uy, *d_uz, *d_pxx, *d_pyy, *d_pzz;
float *d_pxy, *d_pxz, *d_pyz, *d_rho, *d_phi;
float *d_g_out;

// ========================================================================== parametros ========================================================================== //
float H_TAU = 0.505f;
float H_CSSQ = 1.0f / 3.0f;
float H_OMEGA = 1.0f / H_TAU;
float H_SHARP_C = 0.15f * 3.0f;
float H_SIGMA = 0.1f;

// velocity set
#ifdef D3Q19
    int H_CIX[19] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0 };
    int H_CIY[19] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 0, 0, 1, -1, -1, 1, 0, 0, 1, -1 };
    int H_CIZ[19] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, -1, 1, -1, 1 };
#elif defined(D3Q27)
    int H_CIX[27] = { 0, 1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, 1, -1, 1, -1, 1, -1, -1, 1 };
    int H_CIY[27] = { 0, 0, 0, 1, -1, 0, 0, 1, -1, 0, 0, 1, -1, -1, 1, 0, 0, 1, -1, 1, -1, 1, -1, -1, 1, 1, -1 };
    int H_CIZ[27] = { 0, 0, 0, 0, 0, 1, -1, 0, 0, 1, -1, 1, -1, 0, 0, -1, 1, -1, 1, 1, -1, -1, 1, 1, -1, 1, -1 };
#endif

// vs weights
#ifdef D3Q19
    float H_W[19] = {
        1.0f / 3.0f, 
        1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f, 1.0f / 18.0f,
        1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 
        1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f, 1.0f / 36.0f
    };
#elif defined(D3Q27)
    float H_W[27] = {
        8.0f / 27.0f,
        2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 2.0f / 27.0f, 
        1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 1.0f / 54.0f, 
        1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f, 1.0f / 216.0f
    };
#endif

// =============================================================================================================================================================== //

void initializeVars() {
    size_t SIZE = NX * NY * NZ * sizeof(float);            
    size_t DIST_SIZE = NX * NY * NZ * NLINKS * sizeof(float); 

    hipMalloc((void **)&d_rho, SIZE);
    hipMalloc((void **)&d_phi, SIZE);
    hipMalloc((void **)&d_ux, SIZE);
    hipMalloc((void **)&d_uy, SIZE);
    hipMalloc((void **)&d_uz, SIZE);
    hipMalloc((void **)&d_normx, SIZE);
    hipMalloc((void **)&d_normy, SIZE);
    hipMalloc((void **)&d_normz, SIZE);
    hipMalloc((void **)&d_curvature, SIZE);
    hipMalloc((void **)&d_indicator, SIZE);
    hipMalloc((void **)&d_ffx, SIZE);
    hipMalloc((void **)&d_ffy, SIZE);
    hipMalloc((void **)&d_ffz, SIZE);
    hipMalloc((void **)&d_pxx, SIZE);
    hipMalloc((void **)&d_pyy, SIZE);
    hipMalloc((void **)&d_pzz, SIZE);
    hipMalloc((void **)&d_pxy, SIZE);
    hipMalloc((void **)&d_pxz, SIZE);
    hipMalloc((void **)&d_pyz, SIZE);

    hipMalloc((void **)&d_f, DIST_SIZE);
    hipMalloc((void **)&d_g, DIST_SIZE);

    hipMalloc((void **)&d_g_out, DIST_SIZE);

    hipMemset(d_phi, 0, SIZE);
    hipMemset(d_ux, 0, SIZE);
    hipMemset(d_uy, 0, SIZE);
    hipMemset(d_uz, 0, SIZE);
    
    hipMemset(d_f, 0, DIST_SIZE);
    hipMemset(d_g, 0, DIST_SIZE);

    hipMemset(d_normx, 0, SIZE);
    hipMemset(d_normy, 0, SIZE);
    hipMemset(d_normz, 0, SIZE);
    hipMemset(d_curvature, 0, SIZE);
    hipMemset(d_indicator, 0, SIZE);
    hipMemset(d_ffx, 0, SIZE);
    hipMemset(d_ffy, 0, SIZE);
    hipMemset(d_ffz, 0, SIZE);

    hipMemcpyToSymbol(HIP_SYMBOL(CSSQ), &H_CSSQ, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(OMEGA), &H_OMEGA, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(SHARP_C), &H_SHARP_C, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(SIGMA), &H_SIGMA, sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(W), &H_W, NLINKS * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(CIX), &H_CIX, NLINKS * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(CIY), &H_CIY, NLINKS * sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(CIZ), &H_CIZ, NLINKS * sizeof(int));

}

